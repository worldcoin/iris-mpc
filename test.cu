#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>

// Basic CPU impl
void simpleCpuGemm(int m, int n, int k, const int8_t *A, const int8_t *B, int32_t *C)
{
    for (int row = 0; row < m; ++row)
    {
        for (int col = 0; col < n; ++col)
        {
            int32_t sum = 0;
            for (int i = 0; i < k; ++i)
            {
                sum += A[i + row * k] * B[i + col * k];
            }
            C[row + col * m] = sum;
        }
    }
}

int main()
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int m = 99, n = 32, k = 12800;
    std::srand(std::time(0));

    std::vector<int8_t> h_A(m * k);
    std::vector<int8_t> h_B(k * n);
    std::vector<int8_t> h_B_transposed(n * k);
    std::vector<int32_t> h_C(m * n);
    std::vector<int32_t> h_C_cpu(m * n);
    std::vector<int32_t> h_C_cpu_transposed(m * n);

    for (int i = 0; i < m * k; ++i)
    {
        h_A[i] = static_cast<int8_t>(std::rand() % 127);
    }
    for (int i = 0; i < k * n; ++i)
    {
        h_B[i] = static_cast<int8_t>(std::rand() % 127);
    }

    int8_t *A, *B;
    int32_t *C;
    hipMalloc(&A, m * k * sizeof(int8_t));
    hipMalloc(&B, k * n * sizeof(int8_t));
    hipMalloc(&C, m * n * sizeof(int32_t));

    hipMemcpy(A, h_A.data(), m * k * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(B, h_B.data(), k * n * sizeof(int8_t), hipMemcpyHostToDevice);

    const int32_t alpha = 1;
    const int32_t beta = 0;

    // Matmul using cublasGemmEx
    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                 m, n, k,
                 &alpha,
                 A, HIP_R_8I, k,
                 B, HIP_R_8I, k,
                 &beta,
                 C, HIP_R_32I, m,
                 HIPBLAS_COMPUTE_32I_PEDANTIC, HIPBLAS_GEMM_DEFAULT);

    hipMemcpy(h_C.data(), C, m * n * sizeof(int32_t), hipMemcpyDeviceToHost);

    // CPU sanity check
    simpleCpuGemm(m, n, k, h_A.data(), h_B.data(), h_C_cpu.data());

    // Compare the results 
    int diffs = 0;
    for (int i = 0; i < m * n; ++i)
    {
        if (h_C_cpu[i] != h_C[i])
        {
            std::cout << h_C_cpu[i] << " " << h_C[i] << " " << i << "\n";
            diffs++;
        }
    }
    std::cout << "The results " << (diffs == 0 ? "MATCH" : "DO NOT MATCH") << ": " << diffs << " out of " << n * m << " values differ\n";

    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipblasDestroy(handle);

    return 0;
}
