
#include <hip/hip_runtime.h>
#define P 65519

extern "C" __global__ void matmul(int *c, unsigned short *output, unsigned int *a0Sums, unsigned int *a1Sums, int *b0Sums, int *b1Sums, size_t numRows, size_t numElements, size_t numCols, long long lCoeff, unsigned short *rngMasks0, unsigned short *rngMasks1)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements)
    {
        unsigned int a0s = a0Sums[idx % numRows];
        unsigned int a1s = a1Sums[idx % numRows];

        // Correct the sum to unsigned
        int b0s = b0Sums[idx / numRows] + numCols * 128;
        int b1s = b1Sums[idx / numRows] + numCols * 128;

        // Correct the intermediate results to unsigned
        long long c00 = c[idx] + ((a0s + b0s) << 7) - (numCols * 16384);
        long long c01 = c[idx + numElements] + ((a0s + b1s) << 7) - (numCols * 16384);
        long long c10 = c[idx + numElements * 2] + ((a1s + b0s) << 7) - (numCols * 16384);
        long long c11 = c[idx + numElements * 3] + ((a1s + b1s) << 7) - (numCols * 16384);
        unsigned short result = (((c00 + ((c01 + c10) << 8) + (c11 << 16))) * lCoeff) % P;

        output[idx] = ((unsigned int)P + (unsigned int)result + (unsigned int)rngMasks0[idx] - (unsigned int)rngMasks1[idx]) % (unsigned int)P;
    }
}

extern "C" __global__ void reconstructAndCompare(unsigned short *codes_result1, unsigned short *codes_result2, unsigned short *codes_result3, unsigned short *masks_result1, unsigned short *masks_result2, unsigned short *masks_result3, bool *output, double match_ratio, size_t numElements)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements)
    {
        short nom = ((unsigned int)codes_result1[idx] + (unsigned int)codes_result2[idx] + (unsigned int)codes_result3[idx]) % (unsigned int)P;
        short den = ((unsigned int)masks_result1[idx] + (unsigned int)masks_result2[idx] + (unsigned int)masks_result3[idx]) % (unsigned int)P;
        output[idx] = nom > (1 - 2 * match_ratio) * den;
    }
}

extern "C" __global__ void reconstructDebug(unsigned short *codes_result1, unsigned short *codes_result2, unsigned short *codes_result3, unsigned short *masks_result1, unsigned short *masks_result2, unsigned short *masks_result3, double *output1, unsigned short *output2, unsigned short *output3, size_t numElements)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements)
    {
        unsigned short nom = ((unsigned int)codes_result1[idx] + (unsigned int)codes_result2[idx] + (unsigned int)codes_result3[idx]) % (unsigned int)P;
        unsigned short den = ((unsigned int)masks_result1[idx] + (unsigned int)masks_result2[idx] + (unsigned int)masks_result3[idx]) % (unsigned int)P;
        output2[idx] = nom;
        nom = ((unsigned int)nom + 32759) % (unsigned int)P;
        output1[idx] = 0.5 - (double)nom / (2.0 * (double)den) + (32759.0/(2 * (double)den));
        output3[idx] = den;
    }
}